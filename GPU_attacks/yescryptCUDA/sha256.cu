#include "hip/hip_runtime.h"
/**
 * Implementation of the GPU Attack to Yescrypt Password Hashing Scheme (PHS).
 * Based on the Yescrypt Reference Implementation by Alexander Peslyak (Copyright 2013-2015)
 * and Colin Percival (Copyright 2009).
 *
 * Author: The Lyra2 PHC team (http://www.lyra-kdf.net/) -- 2015.
 *
 * This software is hereby placed in the public domain.
 *
 * THIS SOFTWARE IS PROVIDED BY THE AUTHORS ''AS IS'' AND ANY EXPRESS
 * OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE IMPLIED
 * WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE
 * ARE DISCLAIMED. IN NO EVENT SHALL THE AUTHORS OR CONTRIBUTORS BE
 * LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR
 * CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF
 * SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR PROFITS; OR
 * BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY OF LIABILITY,
 * WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT (INCLUDING NEGLIGENCE
 * OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE OF THIS SOFTWARE,
 * EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */

/*-
 * Copyright 2005,2007,2009 Colin Percival
 * All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions
 * are met:
 * 1. Redistributions of source code must retain the above copyright
 *    notice, this list of conditions and the following disclaimer.
 * 2. Redistributions in binary form must reproduce the above copyright
 *    notice, this list of conditions and the following disclaimer in the
 *    documentation and/or other materials provided with the distribution.
 *
 * THIS SOFTWARE IS PROVIDED BY THE AUTHOR AND CONTRIBUTORS ``AS IS'' AND
 * ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE
 * ARE DISCLAIMED.  IN NO EVENT SHALL THE AUTHOR OR CONTRIBUTORS BE LIABLE
 * FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL
 * DAMAGES (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS
 * OR SERVICES; LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION)
 * HOWEVER CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT
 * LIABILITY, OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY
 * OUT OF THE USE OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF
 * SUCH DAMAGE.
 */

#include <sys/types.h>
#include <stdint.h>
#include <string.h>
#include "sysendian.h"
#include "sha256.h"
#include "yescrypt.h"
#include <stdio.h>
#include "gpucommon.h"


__device__ void SHA256_Update_GPU(SHA256_CTX * ctx, const void *in, size_t len, unsigned int totalPasswords);

/*
 * Encode a length len/4 vector of (uint32_t) into a length len vector of
 * (unsigned char) in big-endian form.  Assumes len is a multiple of 4.
 */
__device__ static void be32enc_vect_GPU(unsigned char *dst, const uint32_t *src, size_t len, unsigned int totalPasswords)
{
    unsigned int threadNumber = (blockIdx.x * blockDim.x) + threadIdx.x;
    if (threadNumber < (YESCRYPT_P*totalPasswords)){

        size_t i;

        for (i = 0; i < len / 4; i++)
            be32enc_GPU(dst + i * 4, src[i]);
    }
}

/*
 * Decode a big-endian length len vector of (unsigned char) into a length
 * len/4 vector of (uint32_t).  Assumes len is a multiple of 4.
 */
__device__ static void be32dec_vect_GPU(uint32_t *dst, const unsigned char *src, size_t len, unsigned int totalPasswords)
{
    unsigned int threadNumber = (blockIdx.x * blockDim.x) + threadIdx.x;
    if (threadNumber < (YESCRYPT_P*totalPasswords)){
        size_t i;

        for (i = 0; i < len / 4; i++)
            dst[i] = be32dec_GPU(src + i * 4);
    }
}


/* Elementary functions used by SHA256 */
#define Ch(x, y, z)	((x & (y ^ z)) ^ z)
#define Maj(x, y, z)	((x & (y | z)) | (y & z))
#define SHR(x, n)	(x >> n)
#define ROTR(x, n)	((x >> n) | (x << (32 - n)))
#define S0(x)		(ROTR(x, 2) ^ ROTR(x, 13) ^ ROTR(x, 22))
#define S1(x)		(ROTR(x, 6) ^ ROTR(x, 11) ^ ROTR(x, 25))
#define s0(x)		(ROTR(x, 7) ^ ROTR(x, 18) ^ SHR(x, 3))
#define s1(x)		(ROTR(x, 17) ^ ROTR(x, 19) ^ SHR(x, 10))

/* SHA256 round function */
#define RND(a, b, c, d, e, f, g, h, k)			\
	t0 = h + S1(e) + Ch(e, f, g) + k;		\
	t1 = S0(a) + Maj(a, b, c);			\
	d += t0;					\
	h  = t0 + t1;

/* Adjusted round function for rotating state */
#define RNDr(S, W, i, k)			\
	RND(S[(64 - i) % 8], S[(65 - i) % 8],	\
	    S[(66 - i) % 8], S[(67 - i) % 8],	\
	    S[(68 - i) % 8], S[(69 - i) % 8],	\
	    S[(70 - i) % 8], S[(71 - i) % 8],	\
	    W[i] + k)

/*
 * SHA256 block compression function.  The 256-bit state is transformed via
 * the 512-bit input block to produce a new state.
 */
__device__ static void SHA256_Transform_GPU(uint32_t * state, const unsigned char block[64], unsigned int totalPasswords)
{
    unsigned int threadNumber = (blockIdx.x * blockDim.x) + threadIdx.x;
    if (threadNumber < (YESCRYPT_P*totalPasswords)){

        uint32_t W[64];
        uint32_t S[8];
        uint32_t t0, t1;
        int i;

        /* 1. Prepare message schedule W. */
        be32dec_vect_GPU(W, block, 64, totalPasswords);
        for (i = 16; i < 64; i++)
            W[i] = s1(W[i - 2]) + W[i - 7] + s0(W[i - 15]) + W[i - 16];

        /* 2. Initialize working variables. */
        memcpy(S, state, 32);

        /* 3. Mix. */
        RNDr(S, W, 0, 0x428a2f98);
        RNDr(S, W, 1, 0x71374491);
        RNDr(S, W, 2, 0xb5c0fbcf);
        RNDr(S, W, 3, 0xe9b5dba5);
        RNDr(S, W, 4, 0x3956c25b);
        RNDr(S, W, 5, 0x59f111f1);
        RNDr(S, W, 6, 0x923f82a4);
        RNDr(S, W, 7, 0xab1c5ed5);
        RNDr(S, W, 8, 0xd807aa98);
        RNDr(S, W, 9, 0x12835b01);
        RNDr(S, W, 10, 0x243185be);
        RNDr(S, W, 11, 0x550c7dc3);
        RNDr(S, W, 12, 0x72be5d74);
        RNDr(S, W, 13, 0x80deb1fe);
        RNDr(S, W, 14, 0x9bdc06a7);
        RNDr(S, W, 15, 0xc19bf174);
        RNDr(S, W, 16, 0xe49b69c1);
        RNDr(S, W, 17, 0xefbe4786);
        RNDr(S, W, 18, 0x0fc19dc6);
        RNDr(S, W, 19, 0x240ca1cc);
        RNDr(S, W, 20, 0x2de92c6f);
        RNDr(S, W, 21, 0x4a7484aa);
        RNDr(S, W, 22, 0x5cb0a9dc);
        RNDr(S, W, 23, 0x76f988da);
        RNDr(S, W, 24, 0x983e5152);
        RNDr(S, W, 25, 0xa831c66d);
        RNDr(S, W, 26, 0xb00327c8);
        RNDr(S, W, 27, 0xbf597fc7);
        RNDr(S, W, 28, 0xc6e00bf3);
        RNDr(S, W, 29, 0xd5a79147);
        RNDr(S, W, 30, 0x06ca6351);
        RNDr(S, W, 31, 0x14292967);
        RNDr(S, W, 32, 0x27b70a85);
        RNDr(S, W, 33, 0x2e1b2138);
        RNDr(S, W, 34, 0x4d2c6dfc);
        RNDr(S, W, 35, 0x53380d13);
        RNDr(S, W, 36, 0x650a7354);
        RNDr(S, W, 37, 0x766a0abb);
        RNDr(S, W, 38, 0x81c2c92e);
        RNDr(S, W, 39, 0x92722c85);
        RNDr(S, W, 40, 0xa2bfe8a1);
        RNDr(S, W, 41, 0xa81a664b);
        RNDr(S, W, 42, 0xc24b8b70);
        RNDr(S, W, 43, 0xc76c51a3);
        RNDr(S, W, 44, 0xd192e819);
        RNDr(S, W, 45, 0xd6990624);
        RNDr(S, W, 46, 0xf40e3585);
        RNDr(S, W, 47, 0x106aa070);
        RNDr(S, W, 48, 0x19a4c116);
        RNDr(S, W, 49, 0x1e376c08);
        RNDr(S, W, 50, 0x2748774c);
        RNDr(S, W, 51, 0x34b0bcb5);
        RNDr(S, W, 52, 0x391c0cb3);
        RNDr(S, W, 53, 0x4ed8aa4a);
        RNDr(S, W, 54, 0x5b9cca4f);
        RNDr(S, W, 55, 0x682e6ff3);
        RNDr(S, W, 56, 0x748f82ee);
        RNDr(S, W, 57, 0x78a5636f);
        RNDr(S, W, 58, 0x84c87814);
        RNDr(S, W, 59, 0x8cc70208);
        RNDr(S, W, 60, 0x90befffa);
        RNDr(S, W, 61, 0xa4506ceb);
        RNDr(S, W, 62, 0xbef9a3f7);
        RNDr(S, W, 63, 0xc67178f2);

        /* 4. Mix local working variables into global state */
        for (i = 0; i < 8; i++)
            state[i] += S[i];

        /* Clean the stack. */
        memset(W, 0, 256);
        memset(S, 0, 32);
        t0 = t1 = 0;
    }
}


__device__ static unsigned char PAD_GPU[64] = {
	0x80, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0,
	0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0,
	0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0,
	0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0
};


/* Add padding and terminating bit-count. */
__device__ static void SHA256_Pad_GPU(SHA256_CTX * ctx, unsigned int totalPasswords)
{
    unsigned int threadNumber = (blockIdx.x * blockDim.x) + threadIdx.x;
    if (threadNumber < (YESCRYPT_P*totalPasswords)){

        unsigned char len[8];
        uint32_t r, plen;

        // Convert length to a vector of bytes -- we do this now rather
        // than later because the length will change after we pad.
        be32enc_vect_GPU(len, ctx->count, 8, totalPasswords);

        /* Add 1--64 bytes so that the resulting length is 56 mod 64 */
        r = (ctx->count[1] >> 3) & 0x3f;
        plen = (r < 56) ? (56 - r) : (120 - r);
        SHA256_Update_GPU(ctx, PAD_GPU, (size_t)plen, totalPasswords);

        /* Add the terminating bit-count */
        SHA256_Update_GPU(ctx, len, 8, totalPasswords);
    }
}


/* SHA-256 initialization.  Begins a SHA-256 operation. */
__device__ void SHA256_Init_GPU(SHA256_CTX * ctx, unsigned int totalPasswords)
{
    unsigned int threadNumber = (blockIdx.x * blockDim.x) + threadIdx.x;
    if (threadNumber < (YESCRYPT_P*totalPasswords)){

        /* Zero bits processed so far */
        ctx->count[0] = ctx->count[1] = 0;

        /* Magic initialization constants */
        ctx->state[0] = 0x6A09E667;
        ctx->state[1] = 0xBB67AE85;
        ctx->state[2] = 0x3C6EF372;
        ctx->state[3] = 0xA54FF53A;
        ctx->state[4] = 0x510E527F;
        ctx->state[5] = 0x9B05688C;
        ctx->state[6] = 0x1F83D9AB;
        ctx->state[7] = 0x5BE0CD19;
    }
}


/* Add bytes into the hash */
__device__ void SHA256_Update_GPU(SHA256_CTX * ctx, const void *in, size_t len, unsigned int totalPasswords)
{
    unsigned int threadNumber = (blockIdx.x * blockDim.x) + threadIdx.x;
    if (threadNumber < (YESCRYPT_P*totalPasswords)){

        uint32_t bitlen[2];
        uint32_t r;
        const unsigned char *src = (const unsigned char *)in;

        /* Number of bytes left in the buffer from previous updates */
        r = (ctx->count[1] >> 3) & 0x3f;

        /* Convert the length into a number of bits */
        bitlen[1] = ((uint32_t)len) << 3;
        bitlen[0] = (uint32_t)(len >> 29);

        /* Update number of bits */
        if ((ctx->count[1] += bitlen[1]) < bitlen[1])
            ctx->count[0]++;
        ctx->count[0] += bitlen[0];

        /* Handle the case where we don't need to perform any transforms */
        if (len < 64 - r) {
            memcpy(&ctx->buf[r], src, len);
            return;
        }

        /* Finish the current block */
        memcpy(&ctx->buf[r], src, 64 - r);
        SHA256_Transform_GPU(ctx->state, ctx->buf, totalPasswords);
        src += 64 - r;
        len -= 64 - r;

        /* Perform complete blocks */
        while (len >= 64) {
            SHA256_Transform_GPU(ctx->state, src, totalPasswords);
            src += 64;
            len -= 64;
        }

        /* Copy left over data into buffer */
        memcpy(ctx->buf, src, len);
    }
}

/*
 * SHA-256 finalization.  Pads the input data, exports the hash value,
 * and clears the context state.
 */
__device__ void SHA256_Final_GPU(unsigned char digest[32], SHA256_CTX * ctx, unsigned int totalPasswords)
{
    unsigned int threadNumber = (blockIdx.x * blockDim.x) + threadIdx.x;
    if (threadNumber < (YESCRYPT_P*totalPasswords)){
        /* Add padding */
        SHA256_Pad_GPU(ctx, totalPasswords);

        /* Write the hash */
        be32enc_vect_GPU(digest, ctx->state, 32, totalPasswords);

        /* Clear the context state */
        memset((void *)ctx, 0, sizeof(*ctx));
    }
}


/* Initialize an HMAC-SHA256 operation with the given key. */
__device__ void HMAC_SHA256_Init_GPU(HMAC_SHA256_CTX * ctx, const void * _K, size_t Klen, unsigned int totalPasswords)
{
    unsigned int threadNumber = (blockIdx.x * blockDim.x) + threadIdx.x;
    if (threadNumber < (YESCRYPT_P*totalPasswords)){

        unsigned char pad[64];
        unsigned char khash[32];
        const unsigned char * K = (const unsigned char *)_K;
        size_t i;

        // If Klen > 64, the key is really SHA256(K).
        if (Klen > 64) {
            SHA256_Init_GPU(&ctx->ictx, totalPasswords);
            SHA256_Update_GPU(&ctx->ictx, K, Klen, totalPasswords);
            SHA256_Final_GPU(khash, &ctx->ictx, totalPasswords);
            K = khash;
            Klen = 32;
        }

        // Inner SHA256 operation is SHA256(K xor [block of 0x36] || data).
        SHA256_Init_GPU(&ctx->ictx, totalPasswords);
        memset(pad, 0x36, 64);
        for (i = 0; i < Klen; i++)
            pad[i] ^= K[i];
        SHA256_Update_GPU(&ctx->ictx, pad, 64, totalPasswords);

        // Outer SHA256 operation is SHA256(K xor [block of 0x5c] || hash).
        SHA256_Init_GPU(&ctx->octx, totalPasswords);
        memset(pad, 0x5c, 64);
        for (i = 0; i < Klen; i++)
            pad[i] ^= K[i];
        SHA256_Update_GPU(&ctx->octx, pad, 64, totalPasswords);

        // Clean the stack.
        memset(khash, 0, 32);
	}

}


/* Add bytes to the HMAC-SHA256 operation. */
__device__ void HMAC_SHA256_Update_GPU(HMAC_SHA256_CTX * ctx, const void *in, size_t len, unsigned int totalPasswords)
{
    unsigned int threadNumber = (blockIdx.x * blockDim.x) + threadIdx.x;
    if (threadNumber < (YESCRYPT_P*totalPasswords)){
        /* Feed data to the inner SHA256 operation. */
        SHA256_Update_GPU(&ctx->ictx, in, len, totalPasswords);
    }
}


/* Finish an HMAC-SHA256 operation. */
__device__ void HMAC_SHA256_Final_GPU(unsigned char digest[32], HMAC_SHA256_CTX * ctx, unsigned int totalPasswords)
{
    unsigned int threadNumber = (blockIdx.x * blockDim.x) + threadIdx.x;
    if (threadNumber < (YESCRYPT_P*totalPasswords)){
        unsigned char ihash[32];

        /* Finish the inner SHA256 operation. */
        SHA256_Final_GPU(ihash, &ctx->ictx, totalPasswords);

        /* Feed the inner hash to the outer SHA256 operation. */
        SHA256_Update_GPU(&ctx->octx, ihash, 32, totalPasswords);

        /* Finish the outer SHA256 operation. */
        SHA256_Final_GPU(digest, &ctx->octx, totalPasswords);

        /* Clean the stack. */
        memset(ihash, 0, 32);
    }
}


/**
 * PBKDF2_SHA256(passwd, passwdlen, salt, saltlen, c, buf, dkLen):
 * Compute PBKDF2(passwd, salt, c, dkLen) using HMAC-SHA256 as the PRF, and
 * write the output to buf.  The value dkLen must be at most 32 * (2^32 - 1).
 */
__device__ void PBKDF2_SHA256_GPU(const uint8_t * passwd, size_t passwdlen, const uint8_t * salt,
    size_t saltlen, uint64_t c, uint8_t * buf, size_t dkLen, unsigned int totalPasswords)
{
    unsigned int threadNumber = (blockIdx.x * blockDim.x) + threadIdx.x;
    if (threadNumber < (YESCRYPT_P*totalPasswords)){

        HMAC_SHA256_CTX PShctx, hctx;
        size_t i;
        uint8_t ivec[4];
        uint8_t U[32];
        uint8_t T[32];
        uint64_t j;
        int k;
        size_t clen;

        // Compute HMAC state after processing P and S.
        HMAC_SHA256_Init_GPU(&PShctx, passwd, passwdlen, totalPasswords);
        HMAC_SHA256_Update_GPU(&PShctx, salt, saltlen, totalPasswords);

        // Iterate through the blocks.
        for (i = 0; i * 32 < dkLen; i++) {
        //for (i = 0; i < 1; i++) {
            // Generate INT(i + 1).
            be32enc_GPU(ivec, (uint32_t)(i + 1));

            // Compute U_1 = PRF(P, S || INT(i)).
            memcpy(&hctx, &PShctx, sizeof(HMAC_SHA256_CTX));
            HMAC_SHA256_Update_GPU(&hctx, ivec, 4, totalPasswords);
            HMAC_SHA256_Final_GPU(U, &hctx, totalPasswords);

            // T_i = U_1 ...
            memcpy(T, U, 32);

            for (j = 2; j <= c; j++) {
                // Compute U_j.
                HMAC_SHA256_Init_GPU(&hctx, passwd, passwdlen, totalPasswords);
                HMAC_SHA256_Update_GPU(&hctx, U, 32, totalPasswords);
                HMAC_SHA256_Final_GPU(U, &hctx, totalPasswords);

                // ... xor U_j ...
                for (k = 0; k < 32; k++)
                    T[k] ^= U[k];
            }

            // Copy as many bytes as necessary into buf.
            clen = dkLen - i * 32;
            if (clen > 32)
                clen = 32;
            memcpy(&buf[i*32], T, clen);

        }

        // Clean PShctx, since we never called _Final on it.
        memset(&PShctx, 0, sizeof(HMAC_SHA256_CTX));
    }
}


__global__ void HMAC_SHA256_GPU(uint32_t prehash, uint8_t *passwd_GPU, size_t passwdlen, uint32_t *sha256_GPU, size_t sha_size,
    uint8_t *salt_GPU, size_t saltlen, uint32_t *B_GPU, size_t B_size, unsigned int totalPasswords){

    unsigned int threadNumber = (blockIdx.x * blockDim.x) + threadIdx.x;
    if (threadNumber < (YESCRYPT_P*totalPasswords)){

        uint8_t * newPtr;
        size_t B_totalCells = B_size/sizeof(uint32_t);

        HMAC_SHA256_CTX ctx;

        HMAC_SHA256_Init_GPU(&ctx, "yescrypt-prehash", prehash ? 16 : 8, totalPasswords);

        HMAC_SHA256_Update_GPU(&ctx, &passwd_GPU[threadNumber*passwdlen], passwdlen, totalPasswords);

        HMAC_SHA256_Final_GPU((uint8_t *)&sha256_GPU[8*threadNumber], &ctx, totalPasswords);

        newPtr = (uint8_t *)&sha256_GPU[8*threadNumber];

        /* 1: (B_0 ... B_{p-1}) <-- PBKDF2(P, S, 1, p * MFLen) */
        PBKDF2_SHA256_GPU(&newPtr[0], sha_size, &salt_GPU[saltlen*threadNumber], saltlen, 1, (uint8_t *)&B_GPU[B_totalCells*threadNumber], B_size, totalPasswords);

        blkcpy_GPU((uint32_t *)&sha256_GPU[8*threadNumber], (uint32_t *)&B_GPU[B_totalCells*threadNumber], 8, totalPasswords);
    }
}


__global__ void HMAC_SHA256_GPU_2(uint32_t prehash, uint8_t *passwd_GPU, size_t passwdlen, uint32_t *sha256_GPU, size_t sha_size, uint32_t *B_GPU, size_t B_size,
    uint8_t * buf_GPU, size_t buflen, unsigned int totalPasswords){

    unsigned int threadNumber = (blockIdx.x * blockDim.x) + threadIdx.x;
    if (threadNumber < (YESCRYPT_P*totalPasswords)){

        size_t B_totalCells = B_size/sizeof(uint32_t);
        uint8_t dk[32];
        uint8_t * dkp;

        dkp = &buf_GPU[threadNumber*buflen];

        if (buflen < sha_size) {
            PBKDF2_SHA256_GPU(&passwd_GPU[threadNumber*passwdlen], passwdlen, (uint8_t *)&B_GPU[threadNumber*B_totalCells], B_size, 1, dk, /*sizeof(dk)*/sha_size, totalPasswords);
            dkp = dk;
        }

        // 5: DK <-- PBKDF2(P, B, 1, dkLen)
        PBKDF2_SHA256_GPU(&passwd_GPU[threadNumber*passwdlen], passwdlen, (uint8_t *)&B_GPU[threadNumber*B_totalCells], B_size, 1, &buf_GPU[threadNumber*buflen], buflen, totalPasswords);

        if (!(prehash)) {
            // Compute ClientKey
            {
                HMAC_SHA256_CTX ctx;
                HMAC_SHA256_Init_GPU(&ctx, dkp, sha_size/*sizeof(dk)*/, totalPasswords);
                HMAC_SHA256_Update_GPU(&ctx, "Client Key", 10, totalPasswords);
                HMAC_SHA256_Final_GPU((uint8_t *)&sha256_GPU[threadNumber*8], &ctx, totalPasswords);
            }
            // Compute StoredKey
            {
                SHA256_CTX ctx;
                size_t clen = buflen;
                if (clen > sha_size/*sizeof(dk)*/)
                    clen = sha_size/*sizeof(dk)*/;
                SHA256_Init_GPU(&ctx, totalPasswords);
                SHA256_Update_GPU(&ctx, (uint8_t *)&sha256_GPU[threadNumber*8], sha_size, totalPasswords);
                SHA256_Final_GPU(dk, &ctx, totalPasswords);
                memcpy(&buf_GPU[threadNumber*buflen], dk, clen);
            }
        }
    }
}




